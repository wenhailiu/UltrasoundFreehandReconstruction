#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <cmath>

#include "hip/hip_runtime.h"
#include ""

#include "cuda_declares.h"
#include "BaseImgParams.h"

__constant__ float ImageToVolume_DeviceConstant[640 * 16];

__constant__ int VOL_DIM_X;
__constant__ int VOL_DIM_Y;
__constant__ int VOL_DIM_Z;

__constant__ int US_DIM_X;
__constant__ int US_DIM_Y;
__constant__ int FRAME_NUMBER;

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16
#define BLOCK_SIZE_Z 2

#define GROUPING_SIZE 1

#define INTERP_KERNEL_RADIUS 1
#define HOLEFILLING_KERNEL_RADIUS 1

__device__ int GetPxlIdx(int col_idx, int row_idx, int pag_idx){
    return col_idx + row_idx * blockDim.x * gridDim.x + pag_idx * blockDim.x * gridDim.x * blockDim.y * gridDim.y;
}

__global__ void HoleFilling_GPU(float *Volume_d, float *Volume_cpy_d, float *Weighting_d){
    int col_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int row_idx = threadIdx.y + blockIdx.y * blockDim.y;
    int pag_idx = threadIdx.z + blockIdx.z * blockDim.z;

    int Vol_idx = col_idx + row_idx * VOL_DIM_X + pag_idx * VOL_DIM_X * VOL_DIM_Y; //GetPxlIdx(col_idx, row_idx, pag_idx);

    if(col_idx < VOL_DIM_X && row_idx < VOL_DIM_Y && pag_idx < VOL_DIM_Z){
        if(Weighting_d[Vol_idx] == 0.0f){
            int Acc_NotZeroVoxel = 0;
            float Acc_Sum = 0.0;
            {//Accumulate the not zero voxels:
                for(int k_z = 0 - HOLEFILLING_KERNEL_RADIUS; k_z < 1 + HOLEFILLING_KERNEL_RADIUS; ++k_z){
                    for(int k_y = 0 - HOLEFILLING_KERNEL_RADIUS; k_y < 1 + HOLEFILLING_KERNEL_RADIUS; ++k_y){
                        for(int k_x = 0 - HOLEFILLING_KERNEL_RADIUS; k_x < 1 + HOLEFILLING_KERNEL_RADIUS; ++k_x){
                            //check if the current selected pixel [x, y, z] is out of bound:
                            int Selected_Pxl[3] = {col_idx + k_x, row_idx + k_y, pag_idx + k_z};
                            if( (Selected_Pxl[0] >= 0 && Selected_Pxl[0] < VOL_DIM_X) && 
                                (Selected_Pxl[1] >= 0 && Selected_Pxl[1] < VOL_DIM_Y) && 
                                (Selected_Pxl[2] >= 0 && Selected_Pxl[2] < VOL_DIM_Z) ){
                                //is the current selected pxl ZERO?
                                if( Volume_cpy_d[ Selected_Pxl[0] + Selected_Pxl[1] * VOL_DIM_X + Selected_Pxl[2] * VOL_DIM_X * VOL_DIM_Y ] != 0.0f){ 
                                    //Not Zero, to be selected for hole filling:
                                    Acc_Sum += Volume_cpy_d[ Selected_Pxl[0] + Selected_Pxl[1] * VOL_DIM_X + Selected_Pxl[2] * VOL_DIM_X * VOL_DIM_Y ];
                                    ++Acc_NotZeroVoxel;
                                }
                            }
                        }
                    }
                }
            }
            if(Acc_NotZeroVoxel > 0){
                Volume_d[Vol_idx] = Acc_Sum / Acc_NotZeroVoxel;
            }
        }
    }
}

__device__ void Matrix4x4MultiplyPoint(const float* point_in, float* point_out, int frame_idx){
    float sum_tmp = 0.0f;
    for(int it_row = 0; it_row < 4; ++ it_row){
        for(int it_col = 0; it_col < 4; ++it_col){
            sum_tmp += ImageToVolume_DeviceConstant[frame_idx * 16 + it_row * 4 + it_col] * point_in[it_col];
        }
        point_out[it_row] = sum_tmp;
        sum_tmp = 0.0f;
    }
}

__global__ void US_Distribution_GPU(float* Volume_d, float* Weighting_d, uint8_t* US_Frame_d, 
    //Parameters for Volume:
    // int Vol_Dim_x, int Vol_Dim_y, int Vol_Dim_z, 
    float Vxl_size_x, float Vxl_size_y, float Vxl_size_z, 
    float Vol_Ori_x, float Vol_Ori_y, float Vol_Ori_z 

    //Parameters for Ultrasound frames:
    // int US_Dim_x, int US_Dim_y, int US_FrameNumber
    ){

    int frm_idx = (threadIdx.z + blockIdx.z * blockDim.z) * GROUPING_SIZE;
    int col_idx = (threadIdx.x + blockIdx.x * blockDim.x) * GROUPING_SIZE;
    int row_idx = (threadIdx.y + blockIdx.y * blockDim.y) * GROUPING_SIZE;

    for(int it_frame = 0; it_frame < GROUPING_SIZE; ++it_frame){
        
        for(int it_row = 0; it_row < GROUPING_SIZE; ++it_row){
            
            for(int it_col = 0; it_col < GROUPING_SIZE; ++it_col){
                
                //Iteration starts:
                if( frm_idx < FRAME_NUMBER && row_idx < US_DIM_Y && col_idx < US_DIM_X){
                    //Get US pxl under volume, from pxl to [mm]:
                    float US_pxl[4] = {float(col_idx), float(row_idx), 0.0, 1.0};
                    float US_pxl_Under_Vol[4] = {0.0f};
                    Matrix4x4MultiplyPoint(US_pxl, US_pxl_Under_Vol, frm_idx);

                    // printf("%f, %f, %f \n", US_pxl_Under_Vol[0], US_pxl_Under_Vol[1], US_pxl_Under_Vol[2]);

                    //Derive pxl under volume in [mm] to [VOXEL]:
                    float Vxl_From_US_Pxl[3] = { roundf((US_pxl_Under_Vol[0] - Vol_Ori_x) / Vxl_size_x), roundf((US_pxl_Under_Vol[1] - Vol_Ori_y) / Vxl_size_y), roundf((US_pxl_Under_Vol[2] - Vol_Ori_z) / Vxl_size_z) };
                    
                    //Iterate 6 voxels around:
                    {
                        for(int it_i = 0 - INTERP_KERNEL_RADIUS; it_i < 0 + 1 + INTERP_KERNEL_RADIUS; ++it_i){
                            for(int it_j = 0 - INTERP_KERNEL_RADIUS; it_j < 0 + 1 + INTERP_KERNEL_RADIUS; ++it_j){
                                for(int it_k = 0 - INTERP_KERNEL_RADIUS; it_k < 0 + 1 + INTERP_KERNEL_RADIUS; ++it_k){
                                    int select_x = Vxl_From_US_Pxl[0] + it_i;
                                    int select_y = Vxl_From_US_Pxl[1] + it_j;
                                    int select_z = Vxl_From_US_Pxl[2] + it_k;
                                    if( (select_x < VOL_DIM_X) && (select_x >= 0) &&
                                        (select_y < VOL_DIM_Y) && (select_y >= 0) &&
                                        (select_z < VOL_DIM_Z) && (select_z >= 0)){
                                        //Calculate inverse distance: [voxel] to US [pixel]:
                                        float inv_distance = exp( -sqrt(
                                            ( (select_x * Vxl_size_x + Vol_Ori_x) - US_pxl_Under_Vol[0] ) * 
                                            ( (select_x * Vxl_size_x + Vol_Ori_x) - US_pxl_Under_Vol[0] ) + 
                                            ( (select_y * Vxl_size_y + Vol_Ori_y) - US_pxl_Under_Vol[1] ) * 
                                            ( (select_y * Vxl_size_y + Vol_Ori_y) - US_pxl_Under_Vol[1] ) + 
                                            ( (select_z * Vxl_size_z + Vol_Ori_z) - US_pxl_Under_Vol[2] ) * 
                                            ( (select_z * Vxl_size_z + Vol_Ori_z - US_pxl_Under_Vol[2] ))));
                                        
                                        // float Volume_tmp = 
                                        // printf("%f, %f, %f \n", select_x, select_y, select_z);
                                        
                                        float sum = Volume_d[select_x + select_y * VOL_DIM_X + select_z * VOL_DIM_X * VOL_DIM_Y] * Weighting_d[select_x + select_y * VOL_DIM_X + select_z * VOL_DIM_X * VOL_DIM_Y] + 
                                                    US_Frame_d[col_idx + row_idx * US_DIM_X + frm_idx * US_DIM_X * US_DIM_Y] * inv_distance;
                                        Weighting_d[select_x + select_y * VOL_DIM_X + select_z * VOL_DIM_X * VOL_DIM_Y] += inv_distance;
                                        Volume_d[select_x + select_y * VOL_DIM_X + select_z * VOL_DIM_X * VOL_DIM_Y] = sum / Weighting_d[select_x + select_y * VOL_DIM_X + select_z * VOL_DIM_X * VOL_DIM_Y];
                                    }
                                }
                            }
                        }                
                    }
                }
                ++col_idx;
            }
            ++row_idx;
        }
        ++frm_idx;
    }
    
}

__global__ void GetXY_plane(float *Volume_d, float *plane_d, int Location){
    int col_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int row_idx = threadIdx.y + blockIdx.y * blockDim.y;

    int Vol_x_idx = col_idx;
    int Vol_y_idx = row_idx;
    int Vol_z_idx = Location;

    if(col_idx < VOL_DIM_X && row_idx < VOL_DIM_Y){
        plane_d[col_idx + row_idx * VOL_DIM_X] = Volume_d[Vol_x_idx + Vol_y_idx * VOL_DIM_X + Vol_z_idx * VOL_DIM_X * VOL_DIM_Y];
    }
}

__global__ void GetXZ_plane(float *Volume_d, float *plane_d, int Location){
    int col_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int row_idx = threadIdx.y + blockIdx.y * blockDim.y;

    int Vol_x_idx = col_idx;
    int Vol_y_idx = Location;
    int Vol_z_idx = row_idx;

    if(col_idx < VOL_DIM_X && row_idx < VOL_DIM_Z){
        plane_d[col_idx + row_idx * VOL_DIM_X] = Volume_d[Vol_x_idx + Vol_y_idx * VOL_DIM_X + Vol_z_idx * VOL_DIM_X * VOL_DIM_Y];
    }
}

__global__ void GetYZ_plane(float *Volume_d, float *plane_d, int Location){
    int col_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int row_idx = threadIdx.y + blockIdx.y * blockDim.y;

    int Vol_x_idx = Location;
    int Vol_y_idx = row_idx;
    int Vol_z_idx = col_idx;

    if(col_idx < VOL_DIM_Z && row_idx < VOL_DIM_Y){
        plane_d[col_idx + row_idx * VOL_DIM_Z] = Volume_d[Vol_x_idx + Vol_y_idx * VOL_DIM_X + Vol_z_idx * VOL_DIM_X * VOL_DIM_Y];
    }
}

void DisplayPlane(float *Volume_d, float *plane_d, Ultrasound_Parameters US_Params, Volume_Parameters Vol_Params, int Location, char Axis){
    
    dim3 BlockDim_Plane( 
        BLOCK_SIZE_X, 
        BLOCK_SIZE_Y, 
        1
    );

    switch (Axis)
    {
    case 'Z':
        {
            dim3 GridDim_Plane( 
                int(ceil( float(Vol_Params.dim.x) / BLOCK_SIZE_X )), 
                int(ceil( float(Vol_Params.dim.y) / BLOCK_SIZE_Y )), 
                1
            );
            GetXY_plane<<<GridDim_Plane, BlockDim_Plane>>>(Volume_d, plane_d, Location);
            break;
        }
        
    case 'Y': 
        {   
            dim3 GridDim_Plane( 
                int(ceil( float(Vol_Params.dim.x) / BLOCK_SIZE_X )), 
                int(ceil( float(Vol_Params.dim.z) / BLOCK_SIZE_Y )), 
                1
            );
            GetXZ_plane<<<GridDim_Plane, BlockDim_Plane>>>(Volume_d, plane_d, Location);
            break;
        }
    case 'X': 
        {
            dim3 GridDim_Plane( 
                int(ceil( float(Vol_Params.dim.y) / BLOCK_SIZE_X )), 
                int(ceil( float(Vol_Params.dim.z) / BLOCK_SIZE_Y )), 
                1
            );
            GetYZ_plane<<<GridDim_Plane, BlockDim_Plane>>>(Volume_d, plane_d, Location);
            break;
        }
    default:
        std::cout << "Please enter correct Plane indicator: Z (xy plane), Y (xz plane), X (yz plane). " << std::endl;
        break;
    }
}


void GPU_Setups(bool GPU_Available, Ultrasound_Parameters& US_Params, Volume_Parameters& Vol_Params, const float* TotalMatrices, float* Recon_Volume, float* Weighting_Volume, uint8_t* US_Frames){

    //Asignment for Device constant memory:
    hipMemcpyToSymbol(HIP_SYMBOL(ImageToVolume_DeviceConstant), TotalMatrices, US_Params.FrameNumber * 16 * sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(VOL_DIM_X), &Vol_Params.dim.x, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(VOL_DIM_Y), &Vol_Params.dim.y, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(VOL_DIM_Z), &Vol_Params.dim.z, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(US_DIM_X), &US_Params.dim.x, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(US_DIM_Y), &US_Params.dim.y, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(FRAME_NUMBER), &US_Params.FrameNumber, sizeof(int));

    //Allocate GPU memory:
    float   *Volume_d       = NULL;
    float   *Volume_cpy_d   = NULL;
    float   *Weighting_d    = NULL;
    uint8_t *US_Frame_d     = NULL;

    hipMalloc((void **)&Volume_d, Vol_Params.dim.x * Vol_Params.dim.y * Vol_Params.dim.z * sizeof(float));
    hipMalloc((void **)&Volume_cpy_d, Vol_Params.dim.x * Vol_Params.dim.y * Vol_Params.dim.z * sizeof(float));
    hipMalloc((void **)&Weighting_d, Vol_Params.dim.x * Vol_Params.dim.y * Vol_Params.dim.z * sizeof(float));
    hipMalloc((void **)&US_Frame_d, US_Params.dim.x * US_Params.dim.y * US_Params.FrameNumber * sizeof(uint8_t));

    //Copy mem from host RAM to device RAM:
    hipMemcpy(US_Frame_d, US_Frames, US_Params.dim.x * US_Params.dim.y * US_Params.FrameNumber * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(Weighting_d, Weighting_Volume, Vol_Params.dim.x * Vol_Params.dim.y * Vol_Params.dim.z * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Volume_d, Recon_Volume, Vol_Params.dim.x * Vol_Params.dim.y * Vol_Params.dim.z * sizeof(float), hipMemcpyHostToDevice);

    /*------------------------------------------- Perform Distribution ------------------------------------------ */
    //Define threads distribution:
    dim3 BlockDim_Distribution( 
        BLOCK_SIZE_X, 
        BLOCK_SIZE_Y, 
        BLOCK_SIZE_Z 
    );
    dim3 GridDim_Distribution( 
        int(ceil(float(US_Params.dim.x) / BLOCK_SIZE_X / float(GROUPING_SIZE))), 
        int(ceil(float(US_Params.dim.y) / BLOCK_SIZE_Y / float(GROUPING_SIZE))), 
        int(ceil(float(US_Params.FrameNumber) / BLOCK_SIZE_Z / float(GROUPING_SIZE))) 
    );
    US_Distribution_GPU<<<GridDim_Distribution, BlockDim_Distribution>>>(Volume_d, Weighting_d, US_Frame_d, Vol_Params.pxlSize.x, Vol_Params.pxlSize.y, Vol_Params.pxlSize.z, Vol_Params.orig.x, Vol_Params.orig.y, Vol_Params.orig.z);
    hipMemcpy(Volume_cpy_d, Volume_d, Vol_Params.dim.x * Vol_Params.dim.y * Vol_Params.dim.z * sizeof(float), hipMemcpyDeviceToDevice);
    
    /*----------------------------------------------- Hole Filling ---------------------------------------------- */
    dim3 BlockDim_HoleFilling( 
        BLOCK_SIZE_X, 
        BLOCK_SIZE_Y, 
        BLOCK_SIZE_Z 
    );
    dim3 GridDim_HoleFilling( 
        int(ceil( float(Vol_Params.dim.x) / BLOCK_SIZE_X )), 
        int(ceil( float(Vol_Params.dim.y) / BLOCK_SIZE_Y )), 
        int(ceil( float(Vol_Params.dim.z) / BLOCK_SIZE_Z )) 
    );
    HoleFilling_GPU<<<GridDim_HoleFilling, BlockDim_HoleFilling>>>(Volume_d, Volume_cpy_d, Weighting_d);

    //Complete the remaining memory transfer:
    hipMemcpy(Recon_Volume, Volume_d, Vol_Params.dim.x * Vol_Params.dim.y * Vol_Params.dim.z * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Weighting_Volume, Weighting_d, Vol_Params.dim.x * Vol_Params.dim.y * Vol_Params.dim.z * sizeof(float), hipMemcpyDeviceToHost);

    //For display Image:
    float *plane_XY_d, *plane_XZ_d, *plane_YZ_d;
    hipMalloc((void **)&plane_XY_d, Vol_Params.dim.x * Vol_Params.dim.y * sizeof(float));
    hipMalloc((void **)&plane_XZ_d, Vol_Params.dim.x * Vol_Params.dim.z * sizeof(float));
    hipMalloc((void **)&plane_YZ_d, Vol_Params.dim.y * Vol_Params.dim.z * sizeof(float));

    // DisplayPlane(Volume_d, plane_XY_d, US_Params, Vol_Params, 100, 'Z');

    hipFree(Volume_d);
    hipFree(Volume_cpy_d);
    hipFree(Weighting_d);
    hipFree(US_Frame_d);

    hipFree(plane_XY_d);
    hipFree(plane_XZ_d);
    hipFree(plane_YZ_d);
}